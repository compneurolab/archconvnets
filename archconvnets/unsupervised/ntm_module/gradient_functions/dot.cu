#include "hip/hip_runtime.h"
#define GPU_BUFFER1 gpu_buffers[gpu_ind][buffer_ind1]
#define GPU_BUFFER2 gpu_buffers[gpu_ind][buffer_ind2]
#define GPU_BUFFER_OUT gpu_buffers[gpu_ind][out_buffer_ind]
#define BUFFER_SZ1 buffer_sz[gpu_ind][buffer_ind1]
#define BUFFER_SZ2 buffer_sz[gpu_ind][buffer_ind2]
#define OUT_BUFFER_SZ buffer_sz[gpu_ind][out_buffer_ind]

#define DATA_OUT(A, B) data_out[(A)*buffer2_dim2 + (B)]
#define DATA_OUT_IND(A, B) ((A)*buffer2_dim2 + (B))
#define DATA1(A, B) data1[(A)*buffer1_dim2 + (B)]
#define DATA1_IND(A, B) ((A)*buffer1_dim2 + (B))
#define DATA2(A, B) data2[(A)*buffer2_dim2 + (B)]
#define DATA2_IND(A, B) ((A)*buffer2_dim2 + (B))

#define DATA_OUT_SZ (buffer1_dim1*buffer2_dim2*sizeof(DATA_TYPE))
#define DATA_OUT_NUMEL (buffer1_dim1*buffer2_dim2)

__global__ void dot_kernel(float * data1, float * data2, float * data_out, int buffer1_dim1, int buffer1_dim2, int buffer2_dim1, 
			int buffer2_dim2, int data_out_numel){
	int ind = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	int i,j;
	
	int min_duplicates_per_thread = (int)floor((double)data_out_numel / THREAD_CAPACITY);
	int n_additional_duplicates = data_out_numel % THREAD_CAPACITY;
	
	int n_duplicates = min_duplicates_per_thread;
	if(ind < n_additional_duplicates) n_duplicates++;
	
	int ind_g;
	unsigned data_out_ind, data1_ind, data2_ind;
	for(int dup = 0; dup < n_duplicates; dup++){
		ind_g = dup*THREAD_CAPACITY + ind;
		
		#ifdef DEBUG
		if(ind_g >= data_out_numel) assert(0); // out of bounds
		#endif
		
		i = ind_g / buffer2_dim2;
		j = ind_g % buffer2_dim2;
		
		data_out_ind = DATA_OUT_IND(i,j);
		data1_ind = DATA1_IND(i,0);
		data2_ind = DATA2_IND(0,j);
		
		data_out[data_out_ind] = 0;
		for(int k = 0; k < buffer1_dim2; k++){
			data_out[data_out_ind] += data1[data1_ind] * data2[data2_ind];
			
			data1_ind ++;
			data2_ind += buffer2_dim2;
		}
	}
}

static PyObject *dot(PyObject *self, PyObject *args){
	hipError_t err;
	int gpu_ind, buffer_ind1, buffer_ind2, out_buffer_ind;
	PyTupleObject *buffer_shape1, *buffer_shape2;
	
	if (!PyArg_ParseTuple(args, "iO!iO!ii", &buffer_ind1, &PyTuple_Type, &buffer_shape1, &buffer_ind2, 
			&PyTuple_Type, &buffer_shape2, &out_buffer_ind, &gpu_ind)) 
		return NULL;
        
	if(buffer_ind1 >= N_BUFFERS || buffer_ind1 < 0 || 
			out_buffer_ind >= N_BUFFERS || out_buffer_ind < 0 || 
			buffer_ind2 >= N_BUFFERS || buffer_ind2 < 0){
		printf("buffer index incorrect, set_buffers().\n");
		return NULL;
	}
	
	if(gpu_ind >= N_GPUS || gpu_ind < 0){
		printf("gpu index incorrect, set_buffers().\n");
		return NULL;
	}
	
	if(BUFFER_SZ1 == 0 || BUFFER_SZ2 == 0){
		printf("buffer not initialized. use set_buffers()\n");
		return NULL;
	}
	
	// get sizes
	long buffer1_dim1 = PyLong_AsLong(PyTuple_GetItem((PyObject *)buffer_shape1,0));
	long buffer1_dim2 = PyLong_AsLong(PyTuple_GetItem((PyObject *)buffer_shape1,1));
	
	long buffer2_dim1 = PyLong_AsLong(PyTuple_GetItem((PyObject *)buffer_shape2,0));
	long buffer2_dim2 = PyLong_AsLong(PyTuple_GetItem((PyObject *)buffer_shape2,1));
	
	if(buffer1_dim1*buffer1_dim2*sizeof(DATA_TYPE) != BUFFER_SZ1 || buffer2_dim1*buffer2_dim2*sizeof(DATA_TYPE) != BUFFER_SZ2){
		printf("specified input sizes do not equal to stored gpu buffer. dot_cpu()\n");
		return NULL;
	}
	
	if(OUT_BUFFER_SZ == 0){ // init output buffer
		err = hipMalloc((void**) &GPU_BUFFER_OUT, DATA_OUT_SZ); MALLOC_ERR_CHECK
		
		OUT_BUFFER_SZ = DATA_OUT_SZ;
	}else if(DATA_OUT_SZ != OUT_BUFFER_SZ){ // does the output size match the buffer size?
		printf("output buffer size not allocated to correct size\n");
		return NULL;
	}
	
	hipSetDevice(gpu_ind); CHECK_CUDA_ERR
	
	// determine number of blocks
	int n_blocks = (int)ceil((double)DATA_OUT_NUMEL/MAX_THREADS_PER_BLOCK);
	if(n_blocks >= MAX_BLOCKS) n_blocks = MAX_BLOCKS;
	
	// run kernel
	dot_kernel <<< n_blocks, MAX_THREADS_PER_BLOCK >>> (GPU_BUFFER1, GPU_BUFFER2, GPU_BUFFER_OUT, buffer1_dim1, buffer1_dim2, 
			buffer2_dim1, buffer2_dim2, DATA_OUT_NUMEL);
		
	hipSetDevice(0); CHECK_CUDA_ERR
	
	Py_INCREF(Py_None);
	return Py_None;
}

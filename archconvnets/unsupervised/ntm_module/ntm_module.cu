#define NPY_NO_DEPRECATED_API NPY_1_7_API_VERSION
#include "includes.h"
#include "set_buffer.c"
#include "dot_cpu.cu"
#include "dot_gpu.cu"

static PyMethodDef _ntm_module[] = {
	{"set_buffer", set_buffer, METH_VARARGS},
	{"dot_cpu", dot_cpu, METH_VARARGS},
	{"dot_gpu", dot_gpu, METH_VARARGS},
	{NULL, NULL}
};

extern "C" void init_ntm_module(){
	(void) Py_InitModule("_ntm_module", _ntm_module);
	import_array();
	
	hipError_t err;
	
	/////////////////////////////////////////////////////////
    for(int gpu_ind = 0; gpu_ind < N_GPUS; gpu_ind++){
		hipSetDevice(gpu_ind); CHECK_CUDA_ERR_R
		for(int buffer_ind = 0; buffer_ind < N_BUFFERS; buffer_ind++){
			GPU_BUFFER = NULL;
			BUFFER_SZ = 0;
		}
	}
    
	return;
} 

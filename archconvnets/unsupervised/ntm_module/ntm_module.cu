#define NPY_NO_DEPRECATED_API NPY_1_7_API_VERSION
#include "includes.h"
#include "set_buffer.c"
#include "free_buffer.c"
#include "dot.cu"
#include "return_buffer.c"
#include "sync.c"
#include "cosine_sim_expand_dkeys_cpu.c"
#include "cosine_sim_expand_dmem_cpu.c"
#include "cosine_sim_expand_dkeys.c"
#include "cosine_sim_expand_dmem.c"
#include "softmax_dlayer_in_nsum_cpu.c"
#include "softmax_dlayer_in_nsum.c"
#include "dsharpen_dw_cpu.c"
#include "dsharpen_dw.c"
#include "dsharpen_dgamma_cpu.c"

static PyMethodDef _ntm_module[] = {
	{"sync", sync, METH_VARARGS},
	{"set_buffer", set_buffer, METH_VARARGS},
	{"free_buffer", free_buffer, METH_VARARGS},
	{"dot", dot, METH_VARARGS},
	{"return_buffer", return_buffer, METH_VARARGS},	
	{"cosine_sim_expand_dkeys_cpu", cosine_sim_expand_dkeys_cpu, METH_VARARGS},
	{"cosine_sim_expand_dmem_cpu", cosine_sim_expand_dmem_cpu, METH_VARARGS},
	{"cosine_sim_expand_dkeys", cosine_sim_expand_dkeys, METH_VARARGS},
	{"cosine_sim_expand_dmem", cosine_sim_expand_dmem, METH_VARARGS},
	{"softmax_dlayer_in_nsum_cpu", softmax_dlayer_in_nsum_cpu, METH_VARARGS},
	{"softmax_dlayer_in_nsum", softmax_dlayer_in_nsum, METH_VARARGS},
	{"dsharpen_dw_cpu", dsharpen_dw_cpu, METH_VARARGS},
	{"dsharpen_dw", dsharpen_dw, METH_VARARGS},
	{"dsharpen_dgamma_cpu", dsharpen_dgamma_cpu, METH_VARARGS},
	{NULL, NULL}
};

extern "C" void init_ntm_module(){
	(void) Py_InitModule("_ntm_module", _ntm_module);
	import_array();
	
	/////////////////////////////////////////////////////////
	for(int gpu_ind = 0; gpu_ind < N_GPUS; gpu_ind++){
		for(int buffer_ind = 0; buffer_ind < N_BUFFERS; buffer_ind++){
			GPU_BUFFER = NULL;
			BUFFER_SZ = 0;
		}
	}
    
	return;
} 

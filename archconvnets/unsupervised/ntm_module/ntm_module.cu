#define NPY_NO_DEPRECATED_API NPY_1_7_API_VERSION
#include "includes.h"
#include "set_buffer.c"
#include "free_buffer.c"
#include "return_buffer.c"
#include "sync.c"
#include "gradient_functions/dot.cu"
#include "gradient_functions/cosine_sim_expand_dkeys_cpu.c"
#include "gradient_functions/cosine_sim_expand_dmem_cpu.c"
#include "gradient_functions/cosine_sim_expand_dkeys.c"
#include "gradient_functions/cosine_sim_expand_dmem.c"
#include "gradient_functions/softmax_dlayer_in_cpu.c"
#include "gradient_functions/softmax_dlayer_in.c"
#include "gradient_functions/sharpen_dw_cpu.c"
#include "gradient_functions/sharpen_dw.c"
#include "gradient_functions/sharpen_dgamma_cpu.c"
#include "gradient_functions/sharpen_dgamma.c"

static PyMethodDef _ntm_module[] = {
	{"sync", sync, METH_VARARGS},
	{"set_buffer", set_buffer, METH_VARARGS},
	{"free_buffer", free_buffer, METH_VARARGS},
	{"dot", dot, METH_VARARGS},
	{"return_buffer", return_buffer, METH_VARARGS},	
	{"cosine_sim_expand_dkeys_cpu", cosine_sim_expand_dkeys_cpu, METH_VARARGS},
	{"cosine_sim_expand_dmem_cpu", cosine_sim_expand_dmem_cpu, METH_VARARGS},
	{"cosine_sim_expand_dkeys", cosine_sim_expand_dkeys, METH_VARARGS},
	{"cosine_sim_expand_dmem", cosine_sim_expand_dmem, METH_VARARGS},
	{"softmax_dlayer_in_cpu", softmax_dlayer_in_cpu, METH_VARARGS},
	{"softmax_dlayer_in", softmax_dlayer_in, METH_VARARGS},
	{"sharpen_dw_cpu", sharpen_dw_cpu, METH_VARARGS},
	{"sharpen_dw", sharpen_dw, METH_VARARGS},
	{"sharpen_dgamma_cpu", sharpen_dgamma_cpu, METH_VARARGS},
	{"sharpen_dgamma", sharpen_dgamma, METH_VARARGS},
	{NULL, NULL}
};

extern "C" void init_ntm_module(){
	(void) Py_InitModule("_ntm_module", _ntm_module);
	import_array();
	
	/////////////////////////////////////////////////////////
	for(int gpu_ind = 0; gpu_ind < N_GPUS; gpu_ind++){
		for(int buffer_ind = 0; buffer_ind < N_BUFFERS; buffer_ind++){
			GPU_BUFFER = NULL;
			BUFFER_SZ = 0;
		}
	}
    
	return;
} 

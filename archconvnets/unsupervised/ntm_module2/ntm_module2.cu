#define NPY_NO_DEPRECATED_API NPY_1_7_API_VERSION
#include "includes.h"
#include "set_buffer.c"
#include "free_buffer.c"
#include "return_buffer.c"
#include "sync.c"
#include "return_buffer_sz.c"
#include "gradient_functions/dot.cu"
#include "gradient_functions/linear_F_dF.c"
#include "gradient_functions/linear_F_dx.c"
#include "gradient_functions/sum_points.c"
#include "gradient_functions/sum_points_dinput.c"
#include "gradient_functions/point_wise_add.c"
#include "gradient_functions/add_points_dinput.c"

static PyMethodDef _ntm_module2[] = {
	{"sync", sync, METH_VARARGS},
	{"set_buffer", set_buffer, METH_VARARGS},
	{"free_buffer", free_buffer, METH_VARARGS},
	{"return_buffer", return_buffer, METH_VARARGS},	
	{"return_buffer_sz", return_buffer_sz, METH_VARARGS},
	{"linear_F_dF", linear_F_dF, METH_VARARGS},
	{"linear_F_dx", linear_F_dx, METH_VARARGS},
	{"dot", dot, METH_VARARGS},
	{"sum_points", sum_points, METH_VARARGS},
	{"sum_points_dinput", sum_points_dinput, METH_VARARGS},
	{"point_wise_add", point_wise_add, METH_VARARGS},
	{"add_points_dinput", add_points_dinput, METH_VARARGS},
	{NULL, NULL}
};

extern "C" void init_ntm_module2(){
	(void) Py_InitModule("_ntm_module2", _ntm_module2);
	import_array();
	
	/////////////////////////////////////////////////////////
	for(int gpu_ind = 0; gpu_ind < N_GPUS; gpu_ind++){
		for(int buffer_ind = 0; buffer_ind < N_BUFFERS; buffer_ind++){
			GPU_BUFFER = NULL;
			BUFFER_SZ = 0;
		}
	}
    
	return;
} 

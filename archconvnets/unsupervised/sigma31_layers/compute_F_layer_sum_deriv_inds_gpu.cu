#include "hip/hip_runtime.h"
#define P_IND(A,B)((B) + (A)*(n_inds))

#define F1S_IND(A, B, C, D)((D) + (C)*s1 + (B)*s1*s1 + (A)*s1*s1*3)
#define F2S_IND(A, B, C, D)((D) + (C)*s2 + (B)*s2*s2 + (A)*s2*s2*n1)
#define F3S_IND(A, B, C, D)((D) + (C)*s3 + (B)*s3*s3 + (A)*s3*s3*n2)
#define FLS_IND(A, B, C, D)((D) + (C)*max_output_sz3 + (B)*max_output_sz3_max_output_sz3 + (A)*max_output_sz3_max_output_sz3_n3)

__global__ void kernel_F_layer_sum_deriv_inds(float * F_sum, float * FL321, float * F_partial, float * sigma11, IND_DTYPE * inds, 
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_3,
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1, IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1,
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2, IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2, 
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3_s2, IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3,
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3, IND_DTYPE max_output_sz3_max_output_sz3_s3, IND_DTYPE max_output_sz3_max_output_sz3,
	IND_DTYPE max_output_sz3, int layer_ind, IND_DTYPE n_inds, IND_DTYPE max_output_sz3_max_output_sz3_n3, int N_C,
	int s1, int s2, int s3, int n1, int n2, int n3, int ind_j_stride){
	
	int ind_i = blockIdx.x;
	int ind_j_start = threadIdx.x * ind_j_stride;
	
	if(ind_j_start >= n_inds) return;
	int max_j = ind_j_start + ind_j_stride;
	if(max_j > n_inds){
		max_j = n_inds;
	}
	
	////////////////////////////////////////////// unravel inds
		
	int f1_i = inds[ind_i] / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_3;
	IND_DTYPE r = inds[ind_i] % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_3;
	
	int channel_i = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1;
	
	int a1_x_i = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1;
	
	int a1_y_i = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2;
	
	int f2_i = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2;
	
	int a2_x_i = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2;
	
	int a2_y_i = r / max_output_sz3_max_output_sz3_s3_s3_n3;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3;
	
	int f3_i = r / max_output_sz3_max_output_sz3_s3_s3;
	r = r % max_output_sz3_max_output_sz3_s3_s3;
	
	int a3_x_i = r / max_output_sz3_max_output_sz3_s3;
	r = r % max_output_sz3_max_output_sz3_s3;
	
	int a3_y_i = r / max_output_sz3_max_output_sz3;
	r = r % max_output_sz3_max_output_sz3;
	
	int z1_i = r / (max_output_sz3);
	int z2_i = r % (max_output_sz3);
	
	IND_DTYPE F_sum_ind;
	if(layer_ind == 1){
		F_sum_ind = F1S_IND(f1_i, channel_i, a1_x_i, a1_y_i);
	}else if(layer_ind == 2){
		F_sum_ind = F2S_IND(f2_i, f1_i, a2_x_i, a2_y_i);
	}else if(layer_ind == 3){
		F_sum_ind = F3S_IND(f3_i, f2_i, a3_x_i, a3_y_i);
	}
	
	////////////////////////////////////////////// unravel inds
	int f1_j, channel_j, a1_x_j, a1_y_j, f2_j, a2_x_j, a2_y_j, f3_j, a3_x_j, a3_y_j, z1_j, z2_j;
	int ind_j, cat;
	float temp_sum = 0;
	
	for(ind_j = ind_j_start; ind_j < max_j; ind_j++){
		f1_j = inds[ind_j] / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_3;
		r = inds[ind_j] % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_3;
		
		channel_j = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1;
		r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1;
		
		a1_x_j = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1;
		r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1;
		
		a1_y_j = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2;
		r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2;
		
		f2_j = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2;
		r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2;
		
		a2_x_j = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2;
		r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2;
		
		a2_y_j = r / max_output_sz3_max_output_sz3_s3_s3_n3;
		r = r % max_output_sz3_max_output_sz3_s3_s3_n3;
		
		f3_j = r / max_output_sz3_max_output_sz3_s3_s3;
		r = r % max_output_sz3_max_output_sz3_s3_s3;
		
		a3_x_j = r / max_output_sz3_max_output_sz3_s3;
		r = r % max_output_sz3_max_output_sz3_s3;
		
		a3_y_j = r / max_output_sz3_max_output_sz3;
		r = r % max_output_sz3_max_output_sz3;
		
		z1_j = r / (max_output_sz3);
		z2_j = r % (max_output_sz3);
		
		char matching = 0;
		if(layer_ind == 1 && f1_i == f1_j && channel_i == channel_j && a1_x_i == a1_x_j && a1_y_i == a1_y_j){
			matching = 1;
		}else if(layer_ind == 2 && f2_i == f2_j && f1_i == f1_j && a2_x_i == a2_x_j && a2_y_i == a2_y_j){
			matching = 1;
		}else if(layer_ind == 3 && f3_i == f3_j && f2_i == f2_j && a3_x_i == a3_x_j && a3_y_i == a3_y_j){
			matching = 1;
		}else if(layer_ind == 4 && f3_i == f3_j && z1_i == z1_j && z2_i == z2_j){
			matching = 1;
		}
		
		if(matching == 1){
			for(cat = 0; cat < N_C; cat++){
				if(layer_ind == 4){
					F_sum_ind = FLS_IND(cat, f3_i, z1_i, z2_i);
					atomicAdd(&F_sum[F_sum_ind], FL321[P_IND(cat, ind_i)] * F_partial[P_IND(cat, ind_j)] * sigma11[ind_i + ind_j*n_inds]);
				}else{
					temp_sum += FL321[P_IND(cat, ind_i)] * F_partial[P_IND(cat, ind_j)] * sigma11[ind_i + ind_j*n_inds];
				}
			} // cat
		} // matching
	} //ind_j
	
	if(layer_ind != 4){
		atomicAdd(&F_sum[F_sum_ind], temp_sum);
	}
}

// layer_ind defines which layer to keep
static PyObject *compute_F_layer_sum_deriv_inds_gpu(PyObject *self, PyObject *args){
	hipError_t err;
	PyArrayObject *F1_in, *F2_in, *F3_in, *FL_in, *inds_in, *sigma11_in;
	PyArrayObject *FL321_in, *F_sum_in, *F_partial_in; // F_partial: FL321 sans the layer the deriv. is take wrt
	
	int dims[14];
	int layer_ind;
	IND_DTYPE *inds;
	float *FL321, *F_partial, *F_sum, *sigma11;
	
	if (!PyArg_ParseTuple(args, "O!O!O!O!O!O!O!O!i",  &PyArray_Type, &FL321_in, &PyArray_Type, &F_partial_in, &PyArray_Type, &sigma11_in, 
		&PyArray_Type, &F1_in, &PyArray_Type, &F2_in, &PyArray_Type, &F3_in, &PyArray_Type, &FL_in, 
		&PyArray_Type, &inds_in, &layer_ind)) return NULL;

	if (NULL == FL321_in || NULL == F_partial_in || NULL == sigma11_in ||
		NULL == F1_in || NULL == F2_in || NULL == F3_in || NULL == FL_in)  return NULL;

	inds = (IND_DTYPE *) inds_in -> data;
	FL321 = (float *) FL321_in -> data;
	F_partial = (float *) F_partial_in -> data;
	sigma11 = (float *) sigma11_in -> data;
	
	IND_DTYPE N_C = PyArray_DIM(FL_in, 0);
	IND_DTYPE max_output_sz3 = PyArray_DIM(FL_in, 2);
	IND_DTYPE n3 = PyArray_DIM(F3_in, 0);
	IND_DTYPE n2 = PyArray_DIM(F2_in, 0);
	IND_DTYPE n1 = PyArray_DIM(F1_in, 0);
	IND_DTYPE s1 = PyArray_DIM(F1_in, 2);
	IND_DTYPE s2 = PyArray_DIM(F2_in, 2);
	IND_DTYPE s3 = PyArray_DIM(F3_in, 2);
	IND_DTYPE n_inds = PyArray_DIM(inds_in, 0);
	IND_DTYPE n0 = 3;
	
	if(layer_ind == 1){ // F1 inds
		dims[0] = n1;
		dims[1] = 3;
		dims[2] = s1;
		dims[3] = s1;
	}else if(layer_ind == 2){
		dims[0] = n2;
		dims[1] = n1;
		dims[2] = s2;
		dims[3] = s2;
	}else if(layer_ind == 3){
		dims[0] = n3;
		dims[1] = n2;
		dims[2] = s3;
		dims[3] = s3;
	}else if(layer_ind == 4){
		dims[0] = N_C;
		dims[1] = n3;
		dims[2] = max_output_sz3;
		dims[3] = max_output_sz3;
	}else{
		printf("layer index (%i) not supported\n", layer_ind);
		return NULL;
	}
	
	F_sum_in = (PyArrayObject *) PyArray_FromDims(4, dims, NPY_FLOAT);
	F_sum = (float *) F_sum_in -> data;
	
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_3 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1*s1*3;
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1*s1;
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1;
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2;
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2;
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3_s2 = max_output_sz3*max_output_sz3*s3*s3*n3*s2;
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3_n3 = max_output_sz3*max_output_sz3*s3*s3*n3;
	IND_DTYPE max_output_sz3_max_output_sz3_s3_s3 = max_output_sz3*max_output_sz3*s3*s3;
	IND_DTYPE max_output_sz3_max_output_sz3_s3 = max_output_sz3*max_output_sz3*s3;
	IND_DTYPE max_output_sz3_max_output_sz3 = max_output_sz3*max_output_sz3;
	
	IND_DTYPE max_output_sz3_max_output_sz3_n3 = max_output_sz3*max_output_sz3*n3;
	
	IND_DTYPE F_sum_ind;
	char matching;
	
	//////////// cuda mem
	float * F_sum_c, *FL321_c, *F_partial_c, *sigma11_c;
	IND_DTYPE *inds_c;
	
	hipMalloc((void**) &F_sum_c, dims[0]*dims[1]*dims[2]*dims[3] * DATA_TYPE_SZ); CHECK_CUDA_ERR
	hipMalloc((void**) &FL321_c, N_C*n_inds * DATA_TYPE_SZ); CHECK_CUDA_ERR
	hipMalloc((void**) &F_partial_c, N_C*n_inds * DATA_TYPE_SZ); CHECK_CUDA_ERR
	hipMalloc((void**) &sigma11_c, n_inds*n_inds * DATA_TYPE_SZ); CHECK_CUDA_ERR
	hipMalloc((void**) &inds_c, n_inds * sizeof(IND_DTYPE)); CHECK_CUDA_ERR
	
	hipMemcpy(F_sum_c, F_sum, dims[0]*dims[1]*dims[2]*dims[3]*DATA_TYPE_SZ, hipMemcpyHostToDevice);  CHECK_CUDA_ERR
	hipMemcpy(FL321_c, FL321, N_C*n_inds*DATA_TYPE_SZ, hipMemcpyHostToDevice);  CHECK_CUDA_ERR
	hipMemcpy(F_partial_c, F_partial, N_C*n_inds*DATA_TYPE_SZ, hipMemcpyHostToDevice);  CHECK_CUDA_ERR
	hipMemcpy(sigma11_c, sigma11, n_inds*n_inds*DATA_TYPE_SZ, hipMemcpyHostToDevice);  CHECK_CUDA_ERR
	hipMemcpy(inds_c, inds, n_inds * sizeof(IND_DTYPE), hipMemcpyHostToDevice); CHECK_CUDA_ERR
	
	//////////////
	// can we index directly or do we need to stride?
	int thread_sz;
	int ind_j_stride = 1;
	if(n_inds <= 1024)
		thread_sz = n_inds;
	else{
		thread_sz = 1024;
		ind_j_stride = ceil(n_inds/1024.0);
	}
	printf("%i\n", ind_j_stride);
	///////////////////////////////
	kernel_F_layer_sum_deriv_inds <<<n_inds,thread_sz>>>(F_sum_c, FL321_c, F_partial_c, sigma11_c, inds_c, 
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_3,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2, 
		max_output_sz3_max_output_sz3_s3_s3_n3_s2, max_output_sz3_max_output_sz3_s3_s3_n3,
		max_output_sz3_max_output_sz3_s3_s3, max_output_sz3_max_output_sz3_s3, max_output_sz3_max_output_sz3,
		max_output_sz3, layer_ind, n_inds, max_output_sz3_max_output_sz3_n3, N_C, s1, s2, s3, n1, n2, n3, ind_j_stride);
	
	hipDeviceSynchronize();
	hipMemcpy(F_sum, F_sum_c, dims[0]*dims[1]*dims[2]*dims[3]*DATA_TYPE_SZ, hipMemcpyDeviceToHost);  CHECK_CUDA_ERR
	
	
	hipFree(F_sum_c);
	hipFree(FL321_c);
	hipFree(F_partial_c);
	hipFree(sigma11_c);
	hipFree(inds_c);
	
	return PyArray_Return(F_sum_in);
}

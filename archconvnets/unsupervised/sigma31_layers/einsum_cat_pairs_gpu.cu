#include "hip/hip_runtime.h"
__global__ void kernel(float * sum_res, float * sigma31, float * FL321, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0,
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2,
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2, int max_output_sz3_max_output_sz3_s3_s3_n3_s2, int max_output_sz3_max_output_sz3_s3_s3_n3, int max_output_sz3_max_output_sz3_s3_s3,
	int max_output_sz3_max_output_sz3_s3, int max_output_sz3_max_output_sz3, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s,
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s,
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s, int max_output_sz3_max_output_sz3_s3_s3_n3_s2s, int max_output_sz3_max_output_sz3_s3_s3_n3s, int max_output_sz3_max_output_sz3_s3_s3s,
	int max_output_sz3_max_output_sz3_s3s, int max_output_sz3_max_output_sz3s, int n0, int n0s, int n1, int n1s, int n2, int n2s, int n3, int n3s,
	int max_output_sz3, int max_output_sz3s, int s1, int s1s, int s2, int s2s, int s3, int s3s, int N_C, int FL321_sz){
	int r, s31_ind, cat_ind;
	int cat_i, cat_j;
	int f1, f0;
	int s1x, s1y;
	int f2;
	int s2x, s2y;
	int f3;
	int s3x, s3y;
	int z1, z2;
	
	//----------------------------- init shared mem
	extern __shared__ float sum_res_shared[];
	__syncthreads();
	if(threadIdx.x < (N_C * N_C)){
		sum_res_shared[threadIdx.x] = 0;
	}
	
	int F_ind = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

	if(F_ind >= FL321_sz){
		return;
	}

	r = F_ind;

	cat_j = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1;
	
	f1 = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0;
	
	f0 = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1;
	
	s1x = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1;
	
	s1y = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2;
	
	f2 = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2;
	
	s2x = r / max_output_sz3_max_output_sz3_s3_s3_n3_s2;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3_s2;
	
	s2y = r / max_output_sz3_max_output_sz3_s3_s3_n3;
	r = r % max_output_sz3_max_output_sz3_s3_s3_n3;
	
	f3 = r / max_output_sz3_max_output_sz3_s3_s3;
	r = r % max_output_sz3_max_output_sz3_s3_s3;
	
	s3x = r / max_output_sz3_max_output_sz3_s3;
	r = r % max_output_sz3_max_output_sz3_s3;
	
	s3y = r / max_output_sz3_max_output_sz3;
	r = r % max_output_sz3_max_output_sz3;
	
	z1 = r / max_output_sz3;
	z2 = r % max_output_sz3;
	
	// indices for FL321
	int f1s = 0, f0s = 0;
	int s1xs = 0, s1ys = 0;
	int f2s = 0;
	int s2xs = 0, s2ys = 0;
	int f3s = 0;
	int s3xs = 0, s3ys = 0;
	int z1s = 0, z2s = 0;
	
	// check which dims shouldn't be broadcasted
	if(n1s == n1){
		f1s = f1;
	}
	if(n0s == n0){
		f0s = f0;
	}
	if(s1s == s1){
		s1xs = s1x;
		s1ys = s1y;
	}
	if(n2s == n2){
		f2s = f2;
	}
	if(s2s == s2){
		s2xs = s2x;
		s2ys = s2y;
	}
	if(s3s == s3){
		s3xs = s3x;
		s3ys = s3y;
	}
	if(n3s == n3){
		f3s = f3;
	}
	if(max_output_sz3s == max_output_sz3){
		z1s = z1;
		z2s = z2;
	}
	
	s31_ind = S31_IND(0, f1s, f0s, s1xs, s1ys, f2s, s2xs, s2ys, f3s, s3xs, s3ys, z1s, z2s);
	cat_ind = cat_j*N_C;
	
	for(cat_i = 0; cat_i < N_C; cat_i++){
		atomicAdd(&sum_res_shared[cat_ind], sigma31[s31_ind] * FL321[F_ind]);
		
		cat_ind ++;
		s31_ind += max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s;
	}
	
	__syncthreads();
	if(threadIdx.x < (N_C * N_C)){
		atomicAdd(&sum_res[threadIdx.x], sum_res_shared[threadIdx.x]);
	}
	
}


// inputs: sigma31, FL321
//N_C * n1 * 3 * s1 * s1 * n2 * s2 * s2 * n3 * s3 * s3 * max_output_sz3 * max_output_sz3

/*#define FL321_IND(A,B,C,D,E,F,G,H,I,J,K,L,M)((M) + (L)*max_output_sz3 + (K)*max_output_sz3_max_output_sz3 + (J)*max_output_sz3_max_output_sz3_s3 + (I)*max_output_sz3_max_output_sz3_s3_s3 + \
	(H)*max_output_sz3_max_output_sz3_s3_s3_n3 + (G)*max_output_sz3_max_output_sz3_s3_s3_n3_s2 + (F)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2 + (E)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2 + \
	(D)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1 + (C)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1 + (B)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0 + \
	(A)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1)
	
#define S31_IND(A,B,C,D,E,F,G,H,I,J,K,L,M)((M) + (L)*max_output_sz3s + (K)*max_output_sz3_max_output_sz3s + (J)*max_output_sz3_max_output_sz3_s3s + (I)*max_output_sz3_max_output_sz3_s3_s3s + \
	(H)*max_output_sz3_max_output_sz3_s3_s3_n3s + (G)*max_output_sz3_max_output_sz3_s3_s3_n3_s2s + (F)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s + (E)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s + \
	(D)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s + (C)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s + (B)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s + \
	(A)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s)*/

static PyObject *einsum_cat_pairs_gpu(PyObject *self, PyObject *args){
	PyArrayObject *sigma31_in, *FL321_in;
	hipError_t err;
	PyArrayObject *sum_res_in;
	
	/*if(hipSetDevice(3) != hipSuccess){
		err = hipGetLastError();
		printf("CUDA error: %s\n", hipGetErrorString(err));
		return NULL;
	}*/
	
	float *sigma31, *FL321;
	float *sum_res;
	
	int dims[1];
	
	if (!PyArg_ParseTuple(args, "O!O!", 
		&PyArray_Type, &sigma31_in, &PyArray_Type, &FL321_in)) 
		return NULL;

	if (NULL == sigma31_in || NULL == FL321_in)  return NULL;

	sigma31 = (float *) sigma31_in -> data;
	FL321 = (float *) FL321_in -> data;
	
	//////////////////////// get dims
	
	// dims for FL321
	int N_C = PyArray_DIM(sigma31_in, 0);
	int n1 = PyArray_DIM(FL321_in, 1);
	int n0 = PyArray_DIM(FL321_in, 2);
	int s1 = PyArray_DIM(FL321_in, 3);
	int n2 = PyArray_DIM(FL321_in, 5);
	int s2 = PyArray_DIM(FL321_in, 6);
	int n3 = PyArray_DIM(FL321_in, 8);
	int s3 = PyArray_DIM(FL321_in, 9);
	int max_output_sz3 = PyArray_DIM(FL321_in, 11);

	// dims for sigma
	int n1s = PyArray_DIM(sigma31_in, 1);
	int n0s = PyArray_DIM(sigma31_in, 2);
	int s1s = PyArray_DIM(sigma31_in, 3);
	int n2s = PyArray_DIM(sigma31_in, 5);
	int s2s = PyArray_DIM(sigma31_in, 6);
	int n3s = PyArray_DIM(sigma31_in, 8);
	int s3s = PyArray_DIM(sigma31_in, 9);
	int max_output_sz3s = PyArray_DIM(sigma31_in, 11);

	int FL321_sz = N_C * n1 * n0 * s1 * s1 * n2 * s2 * s2 * n3 * s3 * s3 * max_output_sz3 * max_output_sz3;
	int sigma31_sz = N_C * n1s * n0s * s1s * s1s * n2s * s2s * s2s * n3s * s3s * s3s * max_output_sz3s * max_output_sz3s;
	
	///////////////////////////////// allocate output mem
	dims[0] = N_C * N_C;
	
	sum_res_in = (PyArrayObject *) PyArray_FromDims(1, dims, NPY_FLOAT);
	sum_res = (float *) sum_res_in -> data;
	
	/////////////////////////////////// cuda mem
	float * FL321_c, * sigma31_c;
	float * sum_res_c;
	
	err = hipMalloc((void**) &FL321_c, FL321_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	err = hipMalloc((void**) &sigma31_c, sigma31_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	err = hipMalloc((void**) &sum_res_c, N_C * N_C * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	
	err = hipMemcpy(FL321_c, FL321, FL321_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(sigma31_c, sigma31, sigma31_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(sum_res_c, sum_res, N_C * N_C * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	
	// indexing products
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1*s1*n0*n1;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1*s1*n0;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1*s1;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2 = max_output_sz3*max_output_sz3*s3*s3*n3*s2;
	int max_output_sz3_max_output_sz3_s3_s3_n3 = max_output_sz3*max_output_sz3*s3*s3*n3;
	int max_output_sz3_max_output_sz3_s3_s3 = max_output_sz3*max_output_sz3*s3*s3;
	int max_output_sz3_max_output_sz3_s3 = max_output_sz3*max_output_sz3*s3;
	int max_output_sz3_max_output_sz3 = max_output_sz3*max_output_sz3;
	
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s*s1s*s1s*n0s*n1s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s*s1s*s1s*n0s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s*s1s*s1s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s*s1s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s;
	int max_output_sz3_max_output_sz3_s3_s3_n3s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s;
	int max_output_sz3_max_output_sz3_s3_s3s = max_output_sz3s*max_output_sz3s*s3s*s3s;
	int max_output_sz3_max_output_sz3_s3s = max_output_sz3s*max_output_sz3s*s3s;
	int max_output_sz3_max_output_sz3s = max_output_sz3s*max_output_sz3s;
	
	
	//////////////////////////////////////////////////////////////////////////
	
	dim3 grid_size;
	grid_size.x = ceil(sqrt(FL321_sz / 1024.0));
	grid_size.y = grid_size.x;

	
	kernel<<<grid_size,1024, N_C * N_C * DATA_TYPE_SZ >>>(sum_res_c, sigma31_c, FL321_c, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2, max_output_sz3_max_output_sz3_s3_s3_n3_s2, max_output_sz3_max_output_sz3_s3_s3_n3, max_output_sz3_max_output_sz3_s3_s3,
		max_output_sz3_max_output_sz3_s3, max_output_sz3_max_output_sz3, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s, max_output_sz3_max_output_sz3_s3_s3_n3_s2s, max_output_sz3_max_output_sz3_s3_s3_n3s, max_output_sz3_max_output_sz3_s3_s3s,
		max_output_sz3_max_output_sz3_s3s, max_output_sz3_max_output_sz3s, n0, n0s, n1, n1s, n2, n2s, n3, n3s,
		max_output_sz3, max_output_sz3s, s1, s1s, s2, s2s, s3, s3s, N_C, FL321_sz);
	
	// make the host block until the device is finished with foo
	hipDeviceSynchronize();

	// check for error
	err = hipGetLastError();
	if(err != hipSuccess){
		printf("CUDA error: %s\n", hipGetErrorString(err));
		return NULL;
	}
	
	err = hipMemcpy(sum_res, sum_res_c, N_C * N_C * DATA_TYPE_SZ, hipMemcpyDeviceToHost);  MALLOC_ERR_CHECK
	
	hipFree(FL321_c);
	hipFree(sigma31_c);
	hipFree(sum_res_c);
	
	return PyArray_Return(sum_res_in);
}

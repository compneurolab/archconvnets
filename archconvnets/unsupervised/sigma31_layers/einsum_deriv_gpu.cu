#include "hip/hip_runtime.h"
#define S31_IND2(A,B,C,D,E,F,G,H,I,J,K,L,M)((M)*z2b + (L)*max_output_sz3s + (K)*max_output_sz3_max_output_sz3s + (J)*max_output_sz3_max_output_sz3_s3s + (I)*max_output_sz3_max_output_sz3_s3_s3s + \
	(H)*max_output_sz3_max_output_sz3_s3_s3_n3s + (G)*max_output_sz3_max_output_sz3_s3_s3_n3_s2s + (F)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s + (E)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s + \
	(D)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s + (C)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s + (B)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s + \
	(A)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s)
#define F1_IND(A,B,C,D)(D + (s1)*C + (s1*s1)*B + (s1*s1*n0)*A)
#define F2_IND(A,B,C,D)(D + (s2)*C + (s2*s2)*B + (s2*s2*n1)*A)
#define F3_IND(A,B,C,D)(D + (s3)*C + (s3*s3)*B + (s3*s3*n2)*A)
#define FL_IND(A,B,C,D)(D + (max_output_sz3)*C + (max_output_sz3*max_output_sz3)*B + (max_output_sz3*max_output_sz3*n3)*A)
	
__global__ void kernel_deriv(float * sum_res, float * sigma31, float * F1, float * F2, float * F3, float * FL,
		int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0,
		int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2,
		int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2, int max_output_sz3_max_output_sz3_s3_s3_n3_s2, int max_output_sz3_max_output_sz3_s3_s3_n3, int max_output_sz3_max_output_sz3_s3_s3,
		int max_output_sz3_max_output_sz3_s3, int max_output_sz3_max_output_sz3, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s,
		int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s,
		int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s, int max_output_sz3_max_output_sz3_s3_s3_n3_s2s, int max_output_sz3_max_output_sz3_s3_s3_n3s, int max_output_sz3_max_output_sz3_s3_s3s,
		int max_output_sz3_max_output_sz3_s3s, int max_output_sz3_max_output_sz3s, int z2b, int n0, int n0s, int n1, int n1s, int n2, int n2s, int n3, int n3s,
		int max_output_sz3, int max_output_sz3s, int s1, int s1s, int s2, int s2s, int s3, int s3s, int N_C, int f1_deriv, int f2_deriv, int f3_deriv){
	
	extern __shared__ float sum_res_shared[];
	if(threadIdx.x == 0){
		*sum_res_shared = 0;
	}
	__syncthreads();
	
	int f1, f0;
	int s1x, s1y;
	int f2;
	int s2x, s2y;
	int f3;
	int s3x, s3y;
	int z1, z2;
	int cat_i, cat_j;
	
	////////////////////////////////////////////////////////////////////////////////////////////////////
	// which dimensions have been unraveled across the *grid* and that we should not loop over here? (we are solving for the term containing these particular indices)
	int *f1i = &f1, *f0i = &f0;
	int *s1xi = &s1x, *s1yi = &s1y;
	int *f2i = &f2;
	int *s2xi = &s2x, *s2yi = &s2y;
	int *f3i = &f3;
	int *s3xi = &s3x, *s3yi = &s3y;
	int *z1i = &z1, *z2i = &z2;
	int *cat_ii = &cat_i, *cat_ji = &cat_j;
	
	int f1_sz = n1;
	int f0_sz = n0;
	int s1x_sz = s1;
	int s1y_sz = s1;
	int f2_sz = n2;
	int s2x_sz = s2;
	int s2y_sz = s2;
	int f3_sz = n3;
	int s3x_sz = s3;
	int s3y_sz = s3;
	int z1_sz = max_output_sz3;
	int z2_sz = max_output_sz3;
	int cat_i_sz = N_C;
	int cat_j_sz = N_C;
	
	int r = blockIdx.x;
	if(f1_deriv){
		int cat_jc = r / (N_C*n1*n0*s1*s1);
		cat_ji = &cat_jc;
		r = r % (N_C*n1*n0*s1*s1);
		cat_j_sz = 1;
		
		int cat_ic = r / (n1*n0*s1*s1);
		cat_ii = &cat_ic;
		r = r % (n1*n0*s1*s1);
		cat_i_sz = 1;
	
		int f1c = r / (n0*s1*s1);
		f1i = &f1c;
		r = r % (n0*s1*s1);
		f1_sz = 1;
		
		int f0c = r / (s1*s1);
		f0i = &f0c;
		r = r % (s1*s1);
		f0_sz = 1;
		
		int s1xc = r / s1;
		s1xi = &s1xc;
		s1x_sz = 1;
		
		int s1yc = r % s1;
		s1yi = &s1yc;
		s1y_sz = 1;
	}
	
	float sum_res_local = 0;
	
	//N_C * n1 * 3 * s1 * s1 * n2 * s2 * s2 * n3 * s3 * s3 * max_output_sz3 * max_output_sz3
	for(cat_i = 0; cat_i < cat_i_sz; cat_i++){
		for(cat_j = 0; cat_j < cat_j_sz; cat_j++){
			for(f1 = 0; f1 < f1_sz; f1++){
				for(f0 = 0; f0 < f0_sz; f0++){
					for(s1x = 0; s1x < s1x_sz; s1x++){
						for(s1y = 0; s1y < s1y_sz; s1y++){
							for(f2 = 0; f2 < f2_sz; f2++){
								for(s2x = 0; s2x < s2x_sz; s2x++){
									for(s2y = 0; s2y < s2y_sz; s2y++){
										for(f3 = 0; f3 < f3_sz; f3++){
											for(s3x = 0; s3x < s3x_sz; s3x++){
												for(s3y = 0; s3y < s3y_sz; s3y++){
													for(z1 = 0; z1 < z1_sz; z1++){ 
														for(z2 = 0; z2 < z2_sz; z2++){
															sum_res_local += sigma31[S31_IND2(*cat_ii, *f1i, *f0i, *s1xi, *s1yi, *f2i, *s2xi, *s2yi, *f3i, *s3xi, *s3yi, *z1i, *z2i)] *
																F1[F1_IND(*f1i, *f0i, *s1xi, *s1yi)] * F2[F2_IND(*f2i, *f1i, *s2xi, *s2yi)] * F3[F3_IND(*f3i, *f2i, *s3xi, *s3yi)] * FL[FL_IND(*cat_ji, *f3i, *z1i, *z2i)];
															//sum_res_local += sigma31[S31_IND2(cat_i, *f1i, *f0i, *s1xi, *s1yi, *f2i, *s2xi, *s2yi, *f3i, *s3xi, *s3yi, *z1i, *z2i)] *
															//	F2[F2_IND(*f2i, *f1i, *s2xi, *s2yi)] * F3[F3_IND(*f3i, *f2i, *s3xi, *s3yi)] * FL[FL_IND(cat_j, *f3i, *z1i, *z2i)];
														} // z2
													} // z1
												}
											} // s3x, s3y
										} // f3
									}
								} // s2x, s2y
							} // f2
						}
					} // s1x, s1y
				} // f0
			} // f1
		} // cat_j
	} // cat_i
	atomicAdd(&sum_res_shared[0], sum_res_local);
	
	__syncthreads();
	if(threadIdx.x == 0)
		sum_res[blockIdx.x] = *sum_res_shared;
	//atomicAdd(&sum_res[blockIdx.x], sum_res_local);
}


// inputs: sigma31, FL321
//N_C * n1 * 3 * s1 * s1 * n2 * s2 * s2 * n3 * s3 * s3 * max_output_sz3 * max_output_sz3

static PyObject *einsum_deriv_gpu(PyObject *self, PyObject *args){
	PyArrayObject *sigma31_in, *FL_in, *F3_in, *F2_in, *F1_in;
	hipError_t err;
	PyArrayObject *sum_res_in;
	
	/*if(hipSetDevice(3) != hipSuccess){
		err = hipGetLastError();
		printf("CUDA error: %s\n", hipGetErrorString(err));
		return NULL;
	}*/
	
	float *sigma31, *FL, *F3, *F2, *F1;
	float *sum_res;
	
	int dims[1];
	
	if (!PyArg_ParseTuple(args, "O!O!O!O!O!", 
		&PyArray_Type, &sigma31_in, &PyArray_Type, &F1_in, &PyArray_Type, &F2_in, &PyArray_Type, &F3_in, &PyArray_Type, &FL_in)) 
		return NULL;

	if (NULL == sigma31_in || NULL == F1_in || NULL == F2_in || NULL == F3_in || NULL == FL_in)  return NULL;

	sigma31 = (float *) sigma31_in -> data;
	FL = (float *) FL_in -> data;
	F3 = (float *) F3_in -> data;
	F2 = (float *) F2_in -> data;
	F1 = (float *) F1_in -> data;
	
	//////////////////////// get dims
	
	// dims for FL321
	int N_C = PyArray_DIM(sigma31_in, 0);
	int n1 = PyArray_DIM(F1_in, 0);
	int n0 = PyArray_DIM(F1_in, 1);
	int s1 = PyArray_DIM(F1_in, 2);
	int n2 = PyArray_DIM(F2_in, 0);
	int s2 = PyArray_DIM(F2_in, 2);
	int n3 = PyArray_DIM(F3_in, 0);
	int s3 = PyArray_DIM(F3_in, 2);
	int max_output_sz3 = PyArray_DIM(FL_in, 2);

	// dims for sigma
	int n1s = PyArray_DIM(sigma31_in, 1);
	int n0s = PyArray_DIM(sigma31_in, 2);
	int s1s = PyArray_DIM(sigma31_in, 3);
	int n2s = PyArray_DIM(sigma31_in, 5);
	int s2s = PyArray_DIM(sigma31_in, 6);
	int n3s = PyArray_DIM(sigma31_in, 8);
	int s3s = PyArray_DIM(sigma31_in, 9);
	int max_output_sz3s = PyArray_DIM(sigma31_in, 11);

	int F1_sz = n1 * n0 * s1 * s1;
	int F2_sz = n1 * n2 * s2 * s2;
	int F3_sz = n2 * n3 * s3 * s3;
	int FL_sz = N_C * n3 * max_output_sz3 * max_output_sz3;
	int sigma31_sz = N_C * n1s * n0s * s1s * s1s * n2s * s2s * s2s * n3s * s3s * s3s * max_output_sz3s * max_output_sz3s;
	
	///////////////////////////////// allocate output mem
	int output_sz = N_C * N_C * n1 * n0 * s1 * s1;
	dims[0] = output_sz;
	
	sum_res_in = (PyArrayObject *) PyArray_FromDims(1, dims, NPY_FLOAT);
	sum_res = (float *) sum_res_in -> data;
	
	/////////////////////////////////// cuda mem
	float * FL_c, * F3_c, * F2_c, * F1_c, * sigma31_c;
	float * sum_res_c;
	
	err = hipMalloc((void**) &FL_c, FL_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	err = hipMalloc((void**) &F3_c, F3_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	err = hipMalloc((void**) &F2_c, F2_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	err = hipMalloc((void**) &F1_c, F1_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	err = hipMalloc((void**) &sigma31_c, sigma31_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	err = hipMalloc((void**) &sum_res_c, output_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	
	err = hipMemcpy(FL_c, FL, FL_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(F3_c, F3, F3_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(F2_c, F2, F2_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(F1_c, F1, F1_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	
	err = hipMemcpy(sigma31_c, sigma31, sigma31_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(sum_res_c, sum_res, output_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	
	// indexing products
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1*s1*n0*n1;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1*s1*n0;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1*s1;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2 = max_output_sz3*max_output_sz3*s3*s3*n3*s2;
	int max_output_sz3_max_output_sz3_s3_s3_n3 = max_output_sz3*max_output_sz3*s3*s3*n3;
	int max_output_sz3_max_output_sz3_s3_s3 = max_output_sz3*max_output_sz3*s3*s3;
	int max_output_sz3_max_output_sz3_s3 = max_output_sz3*max_output_sz3*s3;
	int max_output_sz3_max_output_sz3 = max_output_sz3*max_output_sz3;
	
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s*s1s*s1s*n0s*n1s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s*s1s*s1s*n0s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s*s1s*s1s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s*s1s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s;
	int max_output_sz3_max_output_sz3_s3_s3_n3s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s;
	int max_output_sz3_max_output_sz3_s3_s3s = max_output_sz3s*max_output_sz3s*s3s*s3s;
	int max_output_sz3_max_output_sz3_s3s = max_output_sz3s*max_output_sz3s*s3s;
	int max_output_sz3_max_output_sz3s = max_output_sz3s*max_output_sz3s;
	int z2b = 1;
	
	// check which dims should be broadcasted
	if(n1s != n1){
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s = 0;
	}
	if(n0s != n0){
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s = 0;
	}
	if(s1s != s1){
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s = 0;
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s = 0;
	}
	if(n2s != n2){
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s = 0;
	}
	if(s2s != s2){
		max_output_sz3_max_output_sz3_s3_s3_n3s = 0;
		max_output_sz3_max_output_sz3_s3_s3_n3_s2s = 0;
	}
	if(s3s != s3){
		max_output_sz3_max_output_sz3s = 0;
		max_output_sz3_max_output_sz3_s3s = 0;
	}
	if(n3s != n3){
		max_output_sz3_max_output_sz3_s3_s3s = 0;
	}
	if(max_output_sz3s != max_output_sz3){
		max_output_sz3s = 0;
		z2b = 0;
	}
	
	
	//////////////////////////////////////////////////////////////////////////
	
	/*dim3 thread_sz_dim;
	thread_sz_dim.x = thread_sz;
	thread_sz_dim.y = n0;*/
	
	//struct timeval tval_before, tval_after, tval_result;
	//gettimeofday(&tval_before, NULL);
	
	kernel_deriv <<< output_sz, 1, DATA_TYPE_SZ >>> (sum_res_c, sigma31_c, F1_c, F2_c, F3_c, FL_c, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2, max_output_sz3_max_output_sz3_s3_s3_n3_s2, max_output_sz3_max_output_sz3_s3_s3_n3, max_output_sz3_max_output_sz3_s3_s3,
		max_output_sz3_max_output_sz3_s3, max_output_sz3_max_output_sz3, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s, max_output_sz3_max_output_sz3_s3_s3_n3_s2s, max_output_sz3_max_output_sz3_s3_s3_n3s, max_output_sz3_max_output_sz3_s3_s3s,
		max_output_sz3_max_output_sz3_s3s, max_output_sz3_max_output_sz3s, z2b, n0, n0s, n1, n1s, n2, n2s, n3, n3s,
		max_output_sz3, max_output_sz3s, s1, s1s, s2, s2s, s3, s3s, N_C, 1,0,0);
	
	// make the host block until the device is finished with foo
	hipDeviceSynchronize();
	
	/*gettimeofday(&tval_after, NULL);
	timersub(&tval_after, &tval_before, &tval_result);
	printf("Time elapsed: %ld.%06ld\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);*/
	
	// check for error
	err = hipGetLastError();
	if(err != hipSuccess){
		printf("CUDA error: %s\n", hipGetErrorString(err));
		return NULL;
	}
	
	err = hipMemcpy(sum_res, sum_res_c, output_sz * DATA_TYPE_SZ, hipMemcpyDeviceToHost);  MALLOC_ERR_CHECK
	
	hipFree(FL_c);
	hipFree(F3_c);
	hipFree(F2_c);
	hipFree(F1_c);
	hipFree(sigma31_c);
	hipFree(sum_res_c);
	
	return PyArray_Return(sum_res_in);
}

#include "hip/hip_runtime.h"
#define S31_IND2(A,B,C,D,E,F,G,H,I,J,K,L,M)((M)*z2b + (L)*max_output_sz3s + (K)*max_output_sz3_max_output_sz3s + (J)*max_output_sz3_max_output_sz3_s3s + (I)*max_output_sz3_max_output_sz3_s3_s3s + \
	(H)*max_output_sz3_max_output_sz3_s3_s3_n3s + (G)*max_output_sz3_max_output_sz3_s3_s3_n3_s2s + (F)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s + (E)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s + \
	(D)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s + (C)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s + (B)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s + \
	(A)*max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s)
#define F1_IND(A,B,C,D)(D + (s1)*C + (s1*s1)*B + (s1*s1*n0)*A)
#define F2_IND(A,B,C,D)(D + (s2)*C + (s2*s2)*B + (s2*s2*n1)*A)
#define F3_IND(A,B,C,D)(D + (s3)*C + (s3*s3)*B + (s3*s3*n2)*A)
#define FL_IND(A,B,C,D)(D + (max_output_sz3)*C + (max_output_sz3*max_output_sz3)*B + (max_output_sz3*max_output_sz3*n3)*A)
	
__global__ void kernel_deriv(float * sum_res, float * sigma31, float * F1, float * F2, float * F3, float * FL,
		int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0,
		int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2,
		int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2, int max_output_sz3_max_output_sz3_s3_s3_n3_s2, int max_output_sz3_max_output_sz3_s3_s3_n3, int max_output_sz3_max_output_sz3_s3_s3,
		int max_output_sz3_max_output_sz3_s3, int max_output_sz3_max_output_sz3, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s,
		int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s, int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s,
		int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s, int max_output_sz3_max_output_sz3_s3_s3_n3_s2s, int max_output_sz3_max_output_sz3_s3_s3_n3s, int max_output_sz3_max_output_sz3_s3_s3s,
		int max_output_sz3_max_output_sz3_s3s, int max_output_sz3_max_output_sz3s, int z2b, int n0, int n0s, int n1, int n1s, int n2, int n2s, int n3, int n3s,
		int max_output_sz3, int max_output_sz3s, int s1, int s1s, int s2, int s2s, int s3, int s3s, int N_C, int deriv_ind){
	
	extern __shared__ float sum_res_shared[];
	if(threadIdx.x == 0){
		*sum_res_shared = 0;
	}
	__syncthreads();
	
	int f1, f0;
	int s1x, s1y;
	int f2;
	int s2x, s2y;
	int f3;
	int s3x, s3y;
	int z1, z2;
	int cat_i, cat_j;
	
	////////////////////////////////////////////////////////////////////////////////////////////////////
	// which dimensions have been unraveled across the *grid* and that we should not loop over here? (we are solving for the term containing these particular indices)
	int *f1i = &f1, *f0i = &f0;
	int *s1xi = &s1x, *s1yi = &s1y;
	int *f2i = &f2;
	int *s2xi = &s2x, *s2yi = &s2y;
	int *f3i = &f3;
	int *s3xi = &s3x, *s3yi = &s3y;
	int *z1i = &z1, *z2i = &z2;
	int *cat_ii = &cat_i, *cat_ji = &cat_j;
	
	int f1_sz = n1;
	int f0_sz = n0;
	int s1x_sz = s1;
	int s1y_sz = s1;
	int f2_sz = n2;
	int s2x_sz = s2;
	int s2y_sz = s2;
	int f3_sz = n3;
	int s3x_sz = s3;
	int s3y_sz = s3;
	int z1_sz = max_output_sz3;
	int z2_sz = max_output_sz3;
	int cat_i_sz = N_C;
	int cat_j_sz = N_C;
	int output_ind;
	
	int r = blockIdx.x;
	int t = threadIdx.x;
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////// which indices are raveled across the grid and threads?
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////// prediction
	if(deriv_ind == 0){
		///////////////////////////////////////////// indices that we keep (specify output term)
		int cat_jc = r;
		cat_ji = &cat_jc;
		cat_j_sz = 1;
		
		int cat_ic = blockIdx.y;
		cat_ii = &cat_ic;
		cat_i_sz = 1;
	
		output_ind = cat_jc*N_C + cat_ic;
		
		//////////////////////////////////////// indices that are raveled over the threads
		//int f0c = threadIdx.y;
		//f0i = &f0c;
		//f0_sz = 1;
		
		int s1xc = t / (s2*s2*s3);
		s1xi = &s1xc;
		t = t % (s2*s2*s3);
		s1x_sz = 1;
		
		int s2xc = t / (s2*s3);
		s2xi = &s2xc;
		t = t % (s2*s3);
		s2x_sz = 1;
		
		int s2yc = t / s3;
		s2yi = &s2yc;
		s2y_sz = 1;
		
		int s3xc = t % s3;
		s3xi = &s3xc;
		s3x_sz = 1;
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////// F1 deriv
	}else if(deriv_ind == 1){
		///////////////////////////////////////////// indices that we keep (specify output term)
		int cat_jc = r / (N_C*s1*s1);
		cat_ji = &cat_jc;
		r = r % (N_C*s1*s1);
		cat_j_sz = 1;
		
		int cat_ic = r / (s1*s1);
		cat_ii = &cat_ic;
		r = r % (s1*s1);
		cat_i_sz = 1;
	
		int s1xc = r / s1;
		s1xi = &s1xc;
		s1x_sz = 1;
		
		int s1yc = r % s1;
		s1yi = &s1yc;
		s1y_sz = 1;
		
		/////////////////////////////
		int f1c = blockIdx.y;
		f1i = &f1c;
		f1_sz = 1;
		
		int f0c = blockIdx.z;
		f0i = &f0c;
		f0_sz = 1;
		
		output_ind = cat_jc*(N_C*n1*n0*s1*s1) + cat_ic*(n1*n0*s1*s1) + f1c*(n0*s1*s1) + f0c*(s1*s1) + s1xc*s1 + s1yc;
		
		//////////////////////////////////////// indices that are raveled over the threads
		int s2xc = t / (s2*s3*s3);
		s2xi = &s2xc;
		t = t % (s2*s3*s3);
		s2x_sz = 1;
		
		int s2yc = t / (s3*s3);
		s2yi = &s2yc;
		t = t % (s3*s3);
		s2y_sz = 1;
		
		int s3xc = t / s3;
		s3xi = &s3xc;
		s3x_sz = 1;
		
		int s3yc = t % s3;
		s3yi = &s3yc;
		s3y_sz = 1;
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////// F2 deriv
	}else if(deriv_ind == 2){
		///////////////////////////////////////////// indices that we keep (specify output term)
		int cat_jc = r / (N_C*s2*s2);
		cat_ji = &cat_jc;
		r = r % (N_C*s2*s2);
		cat_j_sz = 1;
		
		int cat_ic = r / (s2*s2);
		cat_ii = &cat_ic;
		r = r % (s2*s2);
		cat_i_sz = 1;
	
		int s2xc = r / s2;
		s2xi = &s2xc;
		s2x_sz = 1;
		
		int s2yc = r % s2;
		s2yi = &s2yc;
		s2y_sz = 1;
		
		///////////////////////////////////////
		int f2c = blockIdx.y;
		f2i = &f2c;
		f2_sz = 1;
		
		int f1c = blockIdx.z;
		f1i = &f1c;
		f1_sz = 1;
		
		output_ind = cat_jc*(N_C*n2*n1*s2*s2) + cat_ic*(n2*n1*s2*s2) + f2c*(n1*s2*s2) + f1c*(s2*s2) + s2xc*s2 + s2yc;
		
		//////////////////////////////////////// indices that are raveled over the threads
		int s1xc = t / (s1*s3*s3);
		s1xi = &s1xc;
		t = t % (s1*s3*s3);
		s1x_sz = 1;
		
		int s1yc = t / (s3*s3);
		s1yi = &s1yc;
		t = t % (s3*s3);
		s1y_sz = 1;
		
		int s3xc = t / s3;
		s3xi = &s3xc;
		s3x_sz = 1;
		
		int s3yc = t % s3;
		s3yi = &s3yc;
		s3y_sz = 1;
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////// F3 deriv
	}else if(deriv_ind == 3){
		///////////////////////////////////////////// indices that we keep (specify output term)
		int cat_jc = r / (N_C*s3*s3);
		cat_ji = &cat_jc;
		r = r % (N_C*s3*s3);
		cat_j_sz = 1;
		
		int cat_ic = r / (s3*s3);
		cat_ii = &cat_ic;
		r = r % (s3*s3);
		cat_i_sz = 1;
	
		int s3xc = r / s3;
		s3xi = &s3xc;
		s3x_sz = 1;
		
		int s3yc = r % s3;
		s3yi = &s3yc;
		s3y_sz = 1;
		
		///////////////////////////////////////
		int f3c = blockIdx.y;
		f3i = &f3c;
		f3_sz = 1;
		
		int f2c = blockIdx.z;
		f2i = &f2c;
		f2_sz = 1;
		
		output_ind = cat_jc*(N_C*n3*n2*s3*s3) + cat_ic*(n3*n2*s3*s3) + f3c*(n2*s3*s3) + f2c*(s3*s3) + s3xc*s3 + s3yc;
		
		//////////////////////////////////////// indices that are raveled over the threads
		int s1xc = t / (s1*s2*s2);
		s1xi = &s1xc;
		t = t % (s1*s2*s2);
		s1x_sz = 1;
		
		int s1yc = t / (s2*s2);
		s1yi = &s1yc;
		t = t % (s2*s2);
		s1y_sz = 1;
		
		int s2xc = t / s2;
		s2xi = &s2xc;
		s2x_sz = 1;
		
		int s2yc = t % s2;
		s2yi = &s2yc;
		s2y_sz = 1;
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////// FL deriv
	}else if(deriv_ind == 4){
		///////////////////////////////////////////// indices that we keep (specify output term)
		int cat_ic = r / max_output_sz3;
		cat_ii = &cat_ic;
		cat_i_sz = 1;
		
		int z1c = r % max_output_sz3;
		z1i = &z1c;
		z1_sz = 1;
		
		///////////////////////////////////////
		int z2c = blockIdx.y;
		z2i = &z2c;
		z2_sz = 1;
		
		int f3c = blockIdx.z;
		f3i = &f3c;
		f3_sz = 1;
		
		output_ind = cat_ic*(n3*max_output_sz3*max_output_sz3) + f3c*(max_output_sz3*max_output_sz3) + z1c*max_output_sz3 + z2c;
		
		
		// we want cat_j = cat_i, because we do not need to compute the products sigma31[cat_j] * FL[cat_i] for all cat_i,cat_j (because FL is 1)
		cat_ji = &cat_ic;
		cat_j_sz = 1;
		
		//////////////////////////////////////// indices that are raveled over the threads
		
		
		int s1xc = t / (s1*s2*s2);
		s1xi = &s1xc;
		t = t % (s1*s2*s2);
		s1x_sz = 1;
		
		int s1yc = t / (s2*s2);
		s1yi = &s1yc;
		t = t % (s2*s2);
		s1y_sz = 1;
		
		int s2xc = t / s2;
		s2xi = &s2xc;
		s2x_sz = 1;
		
		int s2yc = t % s2;
		s2yi = &s2yc;
		s2y_sz = 1;
	}
	
	float sum_res_local = 0;
	
	for(cat_i = 0; cat_i < cat_i_sz; cat_i++){
		for(cat_j = 0; cat_j < cat_j_sz; cat_j++){
			for(f1 = 0; f1 < f1_sz; f1++){
				for(f0 = 0; f0 < f0_sz; f0++){
					for(s1x = 0; s1x < s1x_sz; s1x++){
						for(s1y = 0; s1y < s1y_sz; s1y++){
							for(f2 = 0; f2 < f2_sz; f2++){
								for(s2x = 0; s2x < s2x_sz; s2x++){
									for(s2y = 0; s2y < s2y_sz; s2y++){
										for(f3 = 0; f3 < f3_sz; f3++){
											for(s3x = 0; s3x < s3x_sz; s3x++){
												for(s3y = 0; s3y < s3y_sz; s3y++){
													for(z1 = 0; z1 < z1_sz; z1++){ 
														for(z2 = 0; z2 < z2_sz; z2++){
															sum_res_local += sigma31[S31_IND2(*cat_ii, *f1i, *f0i, *s1xi, *s1yi, *f2i, *s2xi, *s2yi, *f3i, *s3xi, *s3yi, *z1i, *z2i)] *
																F1[F1_IND(*f1i, *f0i, *s1xi, *s1yi)] * F2[F2_IND(*f2i, *f1i, *s2xi, *s2yi)] * F3[F3_IND(*f3i, *f2i, *s3xi, *s3yi)] * FL[FL_IND(*cat_ji, *f3i, *z1i, *z2i)];
															//sum_res_local += sigma31[S31_IND2(*cat_ii, *f1i, *f0i, *s1xi, *s1yi, *f2i, *s2xi, *s2yi, *f3i, *s3xi, *s3yi, *z1i, *z2i)] *
															//	F2[F2_IND(*f2i, *f1i, *s2xi, *s2yi)] * F3[F3_IND(*f3i, *f2i, *s3xi, *s3yi)] * FL[FL_IND(*cat_ji, *f3i, *z1i, *z2i)];
														} // z2
													} // z1
												}
											} // s3x, s3y
										} // f3
									}
								} // s2x, s2y
							} // f2
						}
					} // s1x, s1y
				} // f0
			} // f1
		} // cat_j
	} // cat_i

	atomicAdd(&sum_res_shared[0], sum_res_local);
	
	__syncthreads();
	if(threadIdx.x == 0)
		sum_res[output_ind] = *sum_res_shared;
	//atomicAdd(&sum_res[blockIdx.x], sum_res_local);
}


// inputs: sigma31, FL321
//N_C * n1 * 3 * s1 * s1 * n2 * s2 * s2 * n3 * s3 * s3 * max_output_sz3 * max_output_sz3

static PyObject *einsum_deriv_gpu(PyObject *self, PyObject *args){
	PyArrayObject *sigma31_in, *FL_in, *F3_in, *F2_in, *F1_in;
	hipError_t err;
	PyArrayObject *sum_res_in;
	
	/*if(hipSetDevice(3) != hipSuccess){
		err = hipGetLastError();
		printf("CUDA error: %s\n", hipGetErrorString(err));
		return NULL;
	}*/
	
	float *sigma31, *FL, *F3, *F2, *F1;
	float *sum_res;
	
	int deriv_ind;
	int dims[1];
	
	if (!PyArg_ParseTuple(args, "O!O!O!O!O!i", 
		&PyArray_Type, &sigma31_in, &PyArray_Type, &F1_in, &PyArray_Type, &F2_in, &PyArray_Type, &F3_in, &PyArray_Type, &FL_in, &deriv_ind)) 
		return NULL;

	if (NULL == sigma31_in || NULL == F1_in || NULL == F2_in || NULL == F3_in || NULL == FL_in)  return NULL;

	sigma31 = (float *) sigma31_in -> data;
	FL = (float *) FL_in -> data;
	F3 = (float *) F3_in -> data;
	F2 = (float *) F2_in -> data;
	F1 = (float *) F1_in -> data;
	
	//////////////////////// get dims
	
	// dims for FL321
	int N_C = PyArray_DIM(sigma31_in, 0);
	int n1 = PyArray_DIM(F1_in, 0);
	int n0 = PyArray_DIM(F1_in, 1);
	int s1 = PyArray_DIM(F1_in, 2);
	int n2 = PyArray_DIM(F2_in, 0);
	int s2 = PyArray_DIM(F2_in, 2);
	int n3 = PyArray_DIM(F3_in, 0);
	int s3 = PyArray_DIM(F3_in, 2);
	int max_output_sz3 = PyArray_DIM(FL_in, 2);

	// dims for sigma
	int n1s = PyArray_DIM(sigma31_in, 1);
	int n0s = PyArray_DIM(sigma31_in, 2);
	int s1s = PyArray_DIM(sigma31_in, 3);
	int n2s = PyArray_DIM(sigma31_in, 5);
	int s2s = PyArray_DIM(sigma31_in, 6);
	int n3s = PyArray_DIM(sigma31_in, 8);
	int s3s = PyArray_DIM(sigma31_in, 9);
	int max_output_sz3s = PyArray_DIM(sigma31_in, 11);

	int F1_sz = n1 * n0 * s1 * s1;
	int F2_sz = n1 * n2 * s2 * s2;
	int F3_sz = n2 * n3 * s3 * s3;
	int FL_sz = N_C * n3 * max_output_sz3 * max_output_sz3;
	int sigma31_sz = N_C * n1s * n0s * s1s * s1s * n2s * s2s * s2s * n3s * s3s * s3s * max_output_sz3s * max_output_sz3s;
	
	
	////////////////////////////////////////////////////////////////////////// which indices do we unravel across threads?
	int output_sz;
	dim3 thread_sz;
	dim3 grid_sz;

	if(deriv_ind == 0){ // prediction (no deriv)
		thread_sz.x = s1*s2*s2*s3;
		//thread_sz.y = n0;
		output_sz = N_C * N_C;
		grid_sz.x = N_C;
		grid_sz.y = N_C;
	}else if(deriv_ind == 1){ // F1 deriv
		thread_sz.x = s2*s2*s3*s3;
		output_sz = N_C * N_C * n1 * n0 * s1 * s1;
		grid_sz.x = N_C * N_C * s1 * s1;
		grid_sz.y = n1;
		grid_sz.z = n0;
	}else if(deriv_ind == 2){ // F2 deriv
		thread_sz.x = s1*s1*s3*s3;
		output_sz = N_C * N_C * n2 * n1 * s2 * s2;
		grid_sz.x = N_C * N_C * s2 * s2;
		grid_sz.y = n2;
		grid_sz.z = n1;
	}else if(deriv_ind == 3){ // F3 deriv
		thread_sz.x = s1*s1*s2*s2;
		output_sz = N_C * N_C * n3 * n2 * s3 * s3;
		grid_sz.x = N_C * N_C * s3 * s3;
		grid_sz.y = n3;
		grid_sz.z = n2;
	}else if(deriv_ind == 4){ // FL deriv
		thread_sz.x = s1*s1*s2*s2;
		output_sz = N_C * n3 * max_output_sz3 * max_output_sz3;
		grid_sz.x = N_C * max_output_sz3;
		grid_sz.y = max_output_sz3;
		grid_sz.z = n3;
	}
	
	///////////////////////////////// allocate output mem
	dims[0] = output_sz;
	
	sum_res_in = (PyArrayObject *) PyArray_FromDims(1, dims, NPY_FLOAT);
	sum_res = (float *) sum_res_in -> data;
	
	/////////////////////////////////// cuda mem
	float * FL_c, * F3_c, * F2_c, * F1_c, * sigma31_c;
	float * sum_res_c;
	
	err = hipMalloc((void**) &FL_c, FL_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	err = hipMalloc((void**) &F3_c, F3_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	err = hipMalloc((void**) &F2_c, F2_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	err = hipMalloc((void**) &F1_c, F1_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	err = hipMalloc((void**) &sigma31_c, sigma31_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	err = hipMalloc((void**) &sum_res_c, output_sz * DATA_TYPE_SZ); MALLOC_ERR_CHECK
	
	err = hipMemcpy(FL_c, FL, FL_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(F3_c, F3, F3_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(F2_c, F2, F2_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(F1_c, F1, F1_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	
	err = hipMemcpy(sigma31_c, sigma31, sigma31_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(sum_res_c, sum_res, output_sz * DATA_TYPE_SZ, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	
	// indexing products
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1*s1*n0*n1;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1*s1*n0;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1*s1;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2*s1;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2*n2;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2 = max_output_sz3*max_output_sz3*s3*s3*n3*s2*s2;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2 = max_output_sz3*max_output_sz3*s3*s3*n3*s2;
	int max_output_sz3_max_output_sz3_s3_s3_n3 = max_output_sz3*max_output_sz3*s3*s3*n3;
	int max_output_sz3_max_output_sz3_s3_s3 = max_output_sz3*max_output_sz3*s3*s3;
	int max_output_sz3_max_output_sz3_s3 = max_output_sz3*max_output_sz3*s3;
	int max_output_sz3_max_output_sz3 = max_output_sz3*max_output_sz3;
	
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s*s1s*s1s*n0s*n1s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s*s1s*s1s*n0s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s*s1s*s1s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s*s1s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s*n2s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s*s2s;
	int max_output_sz3_max_output_sz3_s3_s3_n3_s2s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s*s2s;
	int max_output_sz3_max_output_sz3_s3_s3_n3s = max_output_sz3s*max_output_sz3s*s3s*s3s*n3s;
	int max_output_sz3_max_output_sz3_s3_s3s = max_output_sz3s*max_output_sz3s*s3s*s3s;
	int max_output_sz3_max_output_sz3_s3s = max_output_sz3s*max_output_sz3s*s3s;
	int max_output_sz3_max_output_sz3s = max_output_sz3s*max_output_sz3s;
	int z2b = 1;
	
	// check which dims should be broadcasted
	if(n1s != n1){
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s = 0;
	}
	if(n0s != n0){
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s = 0;
	}
	if(s1s != s1){
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s = 0;
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s = 0;
	}
	if(n2s != n2){
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s = 0;
	}
	if(s2s != s2){
		max_output_sz3_max_output_sz3_s3_s3_n3s = 0;
		max_output_sz3_max_output_sz3_s3_s3_n3_s2s = 0;
	}
	if(s3s != s3){
		max_output_sz3_max_output_sz3s = 0;
		max_output_sz3_max_output_sz3_s3s = 0;
	}
	if(n3s != n3){
		max_output_sz3_max_output_sz3_s3_s3s = 0;
	}
	if(max_output_sz3s != max_output_sz3){
		max_output_sz3s = 0;
		z2b = 0;
	}
	
	//////////////////////////////////////////////////////////////////////////
	
	/*dim3 thread_sz_dim;
	thread_sz_dim.x = thread_sz;
	thread_sz_dim.y = n0;*/
	
	//struct timeval tval_before, tval_after, tval_result;
	//gettimeofday(&tval_before, NULL);
	
	kernel_deriv <<< grid_sz, thread_sz, DATA_TYPE_SZ >>> (sum_res_c, sigma31_c, F1_c, F2_c, F3_c, FL_c, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2, max_output_sz3_max_output_sz3_s3_s3_n3_s2, max_output_sz3_max_output_sz3_s3_s3_n3, max_output_sz3_max_output_sz3_s3_s3,
		max_output_sz3_max_output_sz3_s3, max_output_sz3_max_output_sz3, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0_n1s, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1_n0s,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1_s1s, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2_s1s, max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2_n2s,
		max_output_sz3_max_output_sz3_s3_s3_n3_s2_s2s, max_output_sz3_max_output_sz3_s3_s3_n3_s2s, max_output_sz3_max_output_sz3_s3_s3_n3s, max_output_sz3_max_output_sz3_s3_s3s,
		max_output_sz3_max_output_sz3_s3s, max_output_sz3_max_output_sz3s, z2b, n0, n0s, n1, n1s, n2, n2s, n3, n3s,
		max_output_sz3, max_output_sz3s, s1, s1s, s2, s2s, s3, s3s, N_C, deriv_ind);
	
	// make the host block until the device is finished with foo
	hipDeviceSynchronize();
	
	/*gettimeofday(&tval_after, NULL);
	timersub(&tval_after, &tval_before, &tval_result);
	printf("Time elapsed: %ld.%06ld\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);*/
	
	// check for error
	err = hipGetLastError();
	if(err != hipSuccess){
		printf("CUDA error: %s\n", hipGetErrorString(err));
		return NULL;
	}
	
	err = hipMemcpy(sum_res, sum_res_c, output_sz * DATA_TYPE_SZ, hipMemcpyDeviceToHost);  MALLOC_ERR_CHECK
	
	hipFree(FL_c);
	hipFree(F3_c);
	hipFree(F2_c);
	hipFree(F1_c);
	hipFree(sigma31_c);
	hipFree(sum_res_c);
	
	return PyArray_Return(sum_res_in);
}

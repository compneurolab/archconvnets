#include "hip/hip_runtime.h"
__global__ void kernel_bp_patch_sigma31_sup(float * deriv, float * sigma_imgs, float * imgs, float * F1, float * F2, float * F3, float * FL, 
	long * output_switches3_x_s31, long * output_switches3_y_s31, long * output_switches2_x_s31, long * output_switches2_y_s31, long * output_switches1_x_s31, long * output_switches1_y_s31,
	long * output_switches3_x, long * output_switches3_y, long * output_switches2_x, long * output_switches2_y, long * output_switches1_x, long * output_switches1_y,
	int N_IMGS, int N_C, int n0, int n1, int n2, int n3, int s1, int s2, int s3, int max_output_sz3, IND_DTYPE max_output_sz3_max_output_sz3,	
	IND_DTYPE max_output_sz3_max_output_sz3_n3, IND_DTYPE max_output_sz2_max_output_sz2, IND_DTYPE max_output_sz2_max_output_sz2_n2, IND_DTYPE max_output_sz1_max_output_sz1,
	IND_DTYPE max_output_sz1_max_output_sz1_n1, IND_DTYPE img_sz_img_sz_3, IND_DTYPE img_sz_img_sz, int deriv_ind, IND_DTYPE max_output_sz2, IND_DTYPE max_output_sz1, float * pred, int img_sz){
	
	int r = blockIdx.x;
	
	int f1, channel, a1_x, a1_y, f2, a2_x, a2_y, f3, a3_x, a3_y, z1, z2, cat;
	int a3_x_global_s31, a3_y_global_s31, a2_x_global_s31, a2_y_global_s31, a1_x_global_s31, a1_y_global_s31;
	
	float F_prod;
	IND_DTYPE deriv_in_ind;
	
	int cat_sz = N_C;
	int f1_sz = n1;
	int f2_sz = n2;
	int f3_sz = n3;
	int channel_sz = 3;
	int a1_x_sz = s1;
	int a1_y_sz = s1;
	int a2_x_sz = s2;
	int a2_y_sz = s2;
	int a3_x_sz = s3;
	int a3_y_sz = s3;
	int z1_sz = max_output_sz3;
	int z2_sz = max_output_sz3;
	
	int * cat_i = &cat;
	int * f1_i = &f1;
	int * f2_i = &f2;
	int * f3_i = &f3;
	int * channel_i = &channel;
	int * a1_x_i = &a1_x;
	int * a1_y_i = &a1_y;
	int * a2_x_i = &a2_x;
	int * a2_y_i = &a2_y;
	int * a3_x_i = &a3_x;
	int * a3_y_i = &a3_y;
	int * z1_i = &z1;
	int * z2_i = &z2;
	
	int a3_y_c = threadIdx.x;
	a3_y_i = &a3_y_c;
	a3_y_sz = 1;
	
	int a3_x_c = threadIdx.y;
	a3_x_i = &a3_x_c;
	a3_x_sz = 1;
	
	deriv_in_ind = r;
	/////////// which loops to unravel across the grid
	if(deriv_ind == 1){
		int f1_c = r / (3*s1*s1);
		r = r % (3*s1*s1);
		f1_i = &f1_c;
		f1_sz = 1;
		
		int channel_c = r / (s1*s1);
		r = r % (s1*s1);
		channel_i = &channel_c;
		channel_sz = 1;
		
		int a1_x_c = r / s1;
		int a1_y_c = r % s1;
		a1_x_i = &a1_x_c;
		a1_y_i = &a1_y_c;
		a1_x_sz = 1;
		a1_y_sz = 1;
	}else if(deriv_ind == 2){
		int f2_c = r / (n1*s2*s2);
		r = r % (n1*s2*s2);
		f2_i = &f2_c;
		f2_sz = 1;
		
		int f1_c = r / (s2*s2);
		r = r % (s2*s2);
		f1_i = &f1_c;
		f1_sz = 1;
		
		int a2_x_c = r / s2;
		int a2_y_c = r % s2;
		a2_x_i = &a2_x_c;
		a2_y_i = &a2_y_c;
		a2_x_sz = 1;
		a2_y_sz = 1;
	}else if(deriv_ind == 3){
		int f3_c = r / (n2*s3*s3);
		r = r % (n2*s3*s3);
		f3_i = &f3_c;
		f3_sz = 1;
		
		int f2_c = r / (s3*s3);
		r = r % (s3*s3);
		f2_i = &f2_c;
		f2_sz = 1;
		
		int a3_x_c = r / s3;
		int a3_y_c = r % s3;
		a3_x_i = &a3_x_c;
		a3_y_i = &a3_y_c;
		a3_x_sz = 1;
		a3_y_sz = 1;
	}else if(deriv_ind == 4){
		int cat_c = r / (n3*max_output_sz3*max_output_sz3);
		r = r % (n3*max_output_sz3*max_output_sz3);
		cat_i = &cat_c;
		cat_sz = 1;
		
		int f3_c = r / (max_output_sz3*max_output_sz3);
		r = r % (max_output_sz3*max_output_sz3);
		f3_i = &f3_c;
		f3_sz = 1;
		
		int z1_c = r / max_output_sz3;
		int z2_c = r % max_output_sz3;
		z1_i = &z1_c;
		z2_i = &z2_c;
		z1_sz = 1;
		z2_sz = 1;
	}
	
	float temp_deriv = 0;
	float F_prod_pred;
	float F32, F321;
	
	int switches_3_ind;
	int switches_2_ind;
	int switches_1_ind;
	
	for(f3=0; f3 < f3_sz; f3++){ for(z1=0; z1 < z1_sz; z1++){ for(z2=0; z2 < z2_sz; z2++){ for(a3_x=0; a3_x < a3_x_sz; a3_x++){ for(a3_y=0; a3_y < a3_y_sz; a3_y++){
		for(f2=0; f2 < f2_sz; f2++){ for(a2_x=0; a2_x < a2_x_sz; a2_x++){ for(a2_y=0; a2_y < a2_y_sz; a2_y++){
			
			F32 = F2[F2_IND(*f2_i, *f1_i, *a2_x_i, *a2_y_i)] * F3[F3_IND(*f3_i, *f2_i, *a3_x_i, *a3_y_i)];

			for(f1=0; f1 < f1_sz; f1++){  for(a1_x=0; a1_x < a1_x_sz; a1_x++){  for(a1_y=0; a1_y < a1_y_sz; a1_y++){ 
			
				F321 = F1[F1_IND(*f1_i, *channel_i, *a1_x_i, *a1_y_i)] * F32;
			
				  for(cat=0; cat < cat_sz; cat++){ 
					switches_3_ind = O3_IND(*cat_i,*f3_i,*z1_i,*z2_i);
					
					F_prod = F321 * FL[switches_3_ind];
					
					//////////////////////////////////////////////// sup
					// pool3 -> conv3
					a3_x_global_s31 = output_switches3_x_s31[switches_3_ind] + *a3_x_i;
					a3_y_global_s31 = output_switches3_y_s31[switches_3_ind] + *a3_y_i;
					
					// pool2 -> conv2
					switches_2_ind = O2_IND(*cat_i,*f2_i,a3_x_global_s31,a3_y_global_s31);
					a2_x_global_s31 = output_switches2_x_s31[switches_2_ind] + *a2_x_i;
					a2_y_global_s31 = output_switches2_y_s31[switches_2_ind] + *a2_y_i;
					
					switches_1_ind = O1_IND(*cat_i,*f1_i,a2_x_global_s31,a2_y_global_s31);
					a1_x_global_s31 = output_switches1_x_s31[switches_1_ind] + *a1_x_i;
					a1_y_global_s31 = output_switches1_y_s31[switches_1_ind] + *a1_y_i;
					
					for(channel=0; channel < channel_sz; channel++){
						temp_deriv -= N_IMGS * F_prod * sigma_imgs[I_IND(*cat_i, *channel_i,a1_x_global_s31,a1_y_global_s31)];
}/*
			}}}}}}}}}}*/
			}
	}}}}}}}}}}} // FL layer
	
	atomicAdd(&deriv[deriv_in_ind], temp_deriv);
	return;
}
// inputs: F1, F2, F3, FL, GPU ind

static PyObject *set_filter_buffers(PyObject *self, PyObject *args){
	PyArrayObject *F1_in, *F2_in, *F3_in, *FL_in;
	hipError_t err;
	
	float *F1, *F2, *F3, *FL;
	
	int gpu_ind;
	
	if (!PyArg_ParseTuple(args, "O!O!O!O!i", 
		&PyArray_Type, &F1_in, &PyArray_Type, &F2_in, &PyArray_Type, &F3_in, &PyArray_Type, &FL_in, &gpu_ind)) 
		return NULL;
	
	if (NULL == F1_in || NULL == F2_in || NULL == F3_in || NULL == FL_in)  return NULL;

	if(gpu_ind >= N_GPUS){
		printf("invalid gpu index\n");
		return NULL;
	}
	
	if(hipSetDevice(gpu_ind) != hipSuccess) CHECK_CUDA_ERR
	
	FL = (float *) FL_in -> data;
	F3 = (float *) F3_in -> data;
	F2 = (float *) F2_in -> data;
	F1 = (float *) F1_in -> data;
	
	int F1_sz = PyArray_NBYTES(F1_in);
	int F2_sz = PyArray_NBYTES(F2_in);
	int F3_sz = PyArray_NBYTES(F3_in);
	int FL_sz = PyArray_NBYTES(FL_in);
	
	/////////////////////////////////// allocate cuda mem
	if(F1s_c[gpu_ind] == 0){
		err = hipMalloc((void**) &F1s_c[gpu_ind], F1_sz); MALLOC_ERR_CHECK
		err = hipMalloc((void**) &F2s_c[gpu_ind], F2_sz); MALLOC_ERR_CHECK
		err = hipMalloc((void**) &F3s_c[gpu_ind], F3_sz); MALLOC_ERR_CHECK
		err = hipMalloc((void**) &FLs_c[gpu_ind], FL_sz); MALLOC_ERR_CHECK
		
		///////////////////////////////// set global dimensions used in the main einsum function
		N_C = PyArray_DIM(FL_in, 0);
		n1 = PyArray_DIM(F1_in, 0);
		n0 = PyArray_DIM(F1_in, 1);
		s1 = PyArray_DIM(F1_in, 2);
		n2 = PyArray_DIM(F2_in, 0);
		s2 = PyArray_DIM(F2_in, 2);
		n3 = PyArray_DIM(F3_in, 0);
		s3 = PyArray_DIM(F3_in, 2);
		max_output_sz3 = PyArray_DIM(FL_in, 2);
		
		
	}else if(N_C != PyArray_DIM(FL_in, 0) || n1 != PyArray_DIM(F1_in, 0) || n0 != PyArray_DIM(F1_in, 1) || 
			s1 != PyArray_DIM(F1_in, 2) || n2 != PyArray_DIM(F2_in, 0) || s2 != PyArray_DIM(F2_in, 2) ||
			n3 != PyArray_DIM(F3_in, 0) || s3 != PyArray_DIM(F3_in, 2) || max_output_sz3 != PyArray_DIM(FL_in, 2)){
				printf("filter dimensions do not match previously stored filter dimensions. they should also be the same across all gpus\n");
				return NULL;
	}
	
	////////////////////////////////// set buffers
	err = hipMemcpy(F1s_c[gpu_ind], F1, F1_sz, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(F2s_c[gpu_ind], F2, F2_sz, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(F3s_c[gpu_ind], F3, F3_sz, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	err = hipMemcpy(FLs_c[gpu_ind], FL, FL_sz, hipMemcpyHostToDevice);  MALLOC_ERR_CHECK
	
	
	CHECK_CUDA_ERR
	
	Py_INCREF(Py_None);
	return Py_None;
}
